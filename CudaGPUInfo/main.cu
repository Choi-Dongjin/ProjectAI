#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

extern "C" __declspec(dllexport) int getCudaGpuInfo() {
	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);

	if (deviceCount == 0)
	{
		printf("No CUDA support device found");
	}

	int devNo = 0;
	hipDeviceProp_t iProp;
	hipGetDeviceProperties(&iProp, devNo);

	printf("   Device %d: %s\n",
		devNo, iProp.name);
	printf("   Number of multiprocessors:           %d\n",
		iProp.multiProcessorCount);
	printf("   clock rate : % d\n",
		iProp.clockRate);
	printf("   Compute capability    :      %d.%d\n",
		iProp.major, iProp.minor);
	printf("   Total amount of global memory :       %4.2f KB\n",
		iProp.totalGlobalMem / 1024.0);
	printf("   Total amount of constant memory:      %4.2f KB\n",
		iProp.totalConstMem / 1024.0);
	printf("   Total amount of shared memory per block:      %4.2f KB\n",
		iProp.sharedMemPerBlock / 1024.0);
	printf("   Total amount of max Threads per block:      %d\n",
		iProp.maxThreadsPerBlock);
	printf("   Total amount of max Grids per x, y, z:      %d %d %d\n",
		iProp.maxGridSize[0], iProp.maxGridSize[1], iProp.maxGridSize[2]);
	printf("   Total amount of max Threads per Dim x, Dim y, DIm z:      %d %d %d\n",
		iProp.maxThreadsDim[0], iProp.maxThreadsDim[1], iProp.maxThreadsDim[2]);
	printf("   Warp size : %d",
		iProp.warpSize);

	int gpuMemory = static_cast<int>(iProp.totalGlobalMem / 1024.0);
	return gpuMemory;
}

int main()
{
	getCudaGpuInfo();
	printf("\nEnd Cuda\n");
}